#include <hip/hip_runtime.h>


#include <math.h>
#include <stdio.h>
#include <stdlib.h>
// #include <chrono>

#ifdef GUI
#include <GL/glut.h>
#include <GL/gl.h>
#include <GL/glu.h>
#endif

#define threshold 0.00001f
#define fire_temp 90.0f
#define wall_temp 0.0f
#define fire_size 20


int block_size = 512; // cuda thread block size
int size; // problem size


__global__ void initialize(float *data, int size) {
    // TODO: intialize the temperature distribution (in parallelized way)
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < (size*size)) {
        data[i] = wall_temp;
    }
}


__global__ void generate_fire_area(bool *fire_area, int size){
    // TODO: generate the fire area (in parallelized way)
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < (size*size)) {
        fire_area[i] = 0;
        float fire1_r2 = fire_size * fire_size;
        int x = i / size;
        int y = i % size;
        int a = x - size / 2;
        int b = y - size / 2;
        int r2 = 0.5 * a * a + 0.8 * b * b - 0.5 * a * b;
        if (r2 < fire1_r2) fire_area[i] = 1;
    }
}


__global__ void update(float *data, float *new_data, int size) {
    // TODO: update temperature for each point  (in parallelized way)
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < (size*size)) {
        if ((i > size) && (i < (size*size-size-1))){
            float up = data[i - size];
            float down = data[i + size];
            float left = data[i - 1];
            float right = data[i + 1];
            float new_val = (up + down + left + right) / 4;
            new_data[i] = new_val;
        }
        
    }
}


__global__ void maintain_wall(float *data, int size) {
    // TODO: maintain the temperature of the wall (sequential is enough)
    
    
}


__global__ void maintain_fire(float *data, bool *fire_area, int size) {
    // TODO: maintain the temperature of the fire (in parallelized way)
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < (size*size)) {
        if (fire_area[i]) data[i] = fire_temp;
    }
}


#ifdef GUI
__global__ void data2pixels(float *data, GLubyte* pixels, int size){
    // TODO: convert rawdata (large, size^2) to pixels (small, size^2) for faster rendering speed (in parallelized way)
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < (size*size)) {
        float temp = data[i];
        int color =  ((int) temp / 5 * 5) * (float) 255 / fire_temp;
        pixels[i * 3] = color;
        pixels[i * 3 + 1] = 255 - color;
        pixels[i * 3 + 2] = 255 - color;
    }
    
}


void plot(GLubyte* pixels){
    // visualize temprature distribution
    #ifdef GUI
    glClear(GL_COLOR_BUFFER_BIT);
    glDrawPixels(size, size, GL_RGB, GL_UNSIGNED_BYTE, pixels);
    glutSwapBuffers();
    #endif
}
#endif


void master() {
    float *data_odd;
    float *data_even;
    bool *fire_area;

    hipMalloc(&data_odd, size * size * sizeof(float));
    hipMalloc(&data_even, size * size * sizeof(float));
    hipMalloc(&fire_area, size * size * sizeof(bool));

    #ifdef GUI
    GLubyte *pixels;
    GLubyte *host_pixels;
    host_pixels = new GLubyte[size * size * 3];
    cudaMalloc(&pixels, size * size * 3 * sizeof(GLubyte));
    #endif

    int n_block_size = size * size / block_size + 1;

    initialize<<<n_block_size, block_size>>>(data_odd, size);
    generate_fire_area<<<n_block_size, block_size>>>(fire_area, size);
    
    int count = 1;
    double total_time = 0;

    while (true){
        // std::chrono::high_size_clock::time_point t1 = std::chrono::high_size_clock::now();

        // TODO: modify the following lines to fit your need.
        if (count % 2 == 1) {
            update<<<n_block_size, block_size>>>(data_odd, data_even, size);
            maintain_fire<<<n_block_size, block_size>>>(data_even, fire_area, size);
            maintain_wall<<<1, 1>>>(data_even, size);
        } else {
            update<<<n_block_size, block_size>>>(data_even, data_odd, size);
            maintain_fire<<<n_block_size, block_size>>>(data_odd, fire_area, size);
            maintain_wall<<<1, 1>>>(data_odd, size);
        }

        // std::chrono::high_size_clock::time_point t2 = std::chrono::high_size_clock::now();
        // double this_time = std::chrono::duration<double>(t2 - t1).count();
        // total_time += this_time;
        // printf("Iteration %d, elapsed time: %.6f\n", count, this_time);
        count++;
        
        #ifdef GUI
        if (count % 2 == 1) {
            data2pixels<<<n_block_size, block_size>>>(data_even, pixels, size);
        } else {
            data2pixels<<<n_block_size, block_size>>>(data_odd, pixels, size);
        }
        cudaMemcpy(host_pixels, pixels, size * size * 3 * sizeof(GLubyte), cudaMemcpyDeviceToHost);
        plot(host_pixels);
        #endif

    }

    printf("Converge after %d iterations, elapsed time: %.6f, average computation time: %.6f\n", count-1, total_time, (double) total_time / (count-1));


    hipFree(data_odd);
    hipFree(data_even);
    hipFree(fire_area);

    #ifdef GUI
    cudaFree(pixels);
    delete[] host_pixels;
    #endif
    
}


int main(int argc, char *argv[]){
    
    size = atoi(argv[1]);

    #ifdef GUI
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(size, size);
    glutCreateWindow("Heat Distribution Simulation Sequential Implementation");
    gluOrtho2D(0, size, 0, size);
    #endif

    master();

    printf("Student ID: 119010001\n"); // replace it with your student id
    printf("Name: Your Name\n"); // replace it with your name
    printf("Assignment 4: Heat Distribution CUDA Implementation\n");

    return 0;

}



